#include "hip/hip_runtime.h"
#include "cpu.h"
#include "iterator.h"
#include <random>

void tensor_fill_cpu(Tensor *a, float value)
{
    for (int32_t i = 0; i < a->size; i++)
    {
        a->data[get_index(a, i)] = value;
    }
}

void tensor_fill_random_uniform_cpu(Tensor *a, float min, float max)
{
    std::mt19937 generator;
    std::uniform_real_distribution<float> uniform(min, max);
    for (int32_t i = 0; i < a->size; i++)
    {
        a->data[get_index(a, i)] = uniform(generator);
    }
}

void tensor_fill_random_normal_cpu(Tensor *a, float mean, float std)
{
    std::mt19937 generator;
    std::normal_distribution<float> normal(mean, std);
    for (int32_t i = 0; i < a->size; i++)
    {
        a->data[get_index(a, i)] = normal(generator);
    }
}

void tensor_fill_identity_cpu(Tensor *a)
{
    int32_t stride_sum = 0;
    for (int32_t i = 0; i < a->dims; i++)
    {
        stride_sum += a->stride[i];
    }
    for (int32_t i = 0; i < a->size; i++)
    {
        int32_t j = get_index(a, i);
        a->data[j] = j % stride_sum == 0 ? 1 : 0;
    }
}

void tensor_unary_minus_cpu(Tensor *a, float *result)
{
    for (int32_t i = 0; i < a->size; i++)
    {
        result[i] = -a->data[get_index(a, i)];
    }
}

void tensor_add_cpu(Tensor *a, Tensor *b, float *result)
{
    for (int32_t i = 0; i < a->size; i++)
    {
        result[i] = a->data[get_index(a, i)] + b->data[get_index(b, i)];
    }
}

void tensor_subtract_cpu(Tensor *a, Tensor *b, float *result)
{
    for (int32_t i = 0; i < a->size; i++)
    {
        result[i] = a->data[get_index(a, i)] - b->data[get_index(b, i)];
    }
}

void tensor_multiply_cpu(Tensor *a, Tensor *b, float *result)
{
    for (int32_t i = 0; i < a->size; i++)
    {
        result[i] = a->data[get_index(a, i)] * b->data[get_index(b, i)];
    }
}

void tensor_divide_cpu(Tensor *a, Tensor *b, float *result)
{
    for (int32_t i = 0; i < a->size; i++)
    {
        result[i] = a->data[get_index(a, i)] / b->data[get_index(b, i)];
    }
}

// TxMxP @ TxPxN => TxMxN
void tensor_matmul_cpu(Tensor *a, Tensor *b, int32_t batch, float *result)
{
    int32_t res_height = a->shape[a->dims - 2];
    int32_t res_width = b->shape[b->dims - 1];
    int32_t common_dim = a->shape[a->dims - 1];
    int32_t a_idx, b_idx;

    int32_t a_batch_stride = res_height * common_dim;
    int32_t b_batch_stride = common_dim * res_width;
    int32_t r_batch_stride = res_height * res_width;

    for (int32_t t = 0; t < batch; t++)
    {
        for (int32_t i = 0; i < res_height; i++)
        {
            for (int32_t j = 0; j < res_width; j++)
            {
                float tmp = 0;
                for (int32_t k = 0; k < common_dim; k++)
                {
                    a_idx = t * a_batch_stride + i * a->stride[a->dims - 2] + k * a->stride[a->dims - 1];
                    b_idx = t * b_batch_stride + k * b->stride[b->dims - 2] + j * b->stride[b->dims - 1];
                    tmp += a->data[get_index(a, a_idx)] * b->data[get_index(b, b_idx)];
                }
                result[t * r_batch_stride + i * res_width + j] = tmp;
            }
        }
    }
}

void tensor_broadcast_add_cpu(Tensor *a, float value, float *result)
{
    for (int32_t i = 0; i < a->size; i++)
    {
        result[i] = a->data[get_index(a, i)] + value;
    }
}

void tensor_broadcast_subtract_cpu(Tensor *a, float value, float *result)
{
    for (int32_t i = 0; i < a->size; i++)
    {
        result[i] = a->data[get_index(a, i)] - value;
    }
}

void tensor_broadcast_multiply_cpu(Tensor *a, float value, float *result)
{
    for (int32_t i = 0; i < a->size; i++)
    {
        result[i] = a->data[get_index(a, i)] * value;
    }
}

void tensor_broadcast_divide_cpu(Tensor *a, float value, float *result)
{
    for (int32_t i = 0; i < a->size; i++)
    {
        result[i] = a->data[get_index(a, i)] / value;
    }
}

void tensor_broadcast_right_divide_cpu(Tensor *a, float value, float *result)
{
    for (int32_t i = 0; i < a->size; i++)
    {
        result[i] = value / a->data[get_index(a, i)];
    }
}

void tensor_power_cpu(Tensor *a, float power, float *result)
{
    for (int32_t i = 0; i < a->size; i++)
    {
        result[i] = pow(a->data[get_index(a, i)], power);
    }
}

void tensor_exp_cpu(Tensor *a, float *result)
{
    for (int32_t i = 0; i < a->size; i++)
    {
        result[i] = exp(a->data[get_index(a, i)]);
    }
}

void tensor_log_cpu(Tensor *a, float *result)
{
    for (int32_t i = 0; i < a->size; i++)
    {
        result[i] = log(a->data[get_index(a, i)]);
    }
}

void tensor_log10_cpu(Tensor *a, float *result)
{
    for (int32_t i = 0; i < a->size; i++)
    {
        result[i] = log10(a->data[get_index(a, i)]);
    }
}
void tensor_logb_cpu(Tensor *a, float base, float *result)
{
    float inverse_log_base = 1.0 / log(base);
    for (int32_t i = 0; i < a->size; i++)
    {
        result[i] = log(a->data[get_index(a, i)]) * inverse_log_base;
    }
}

void tensor_sin_cpu(Tensor *a, float *result)
{
    for (int32_t i = 0; i < a->size; i++)
    {
        result[i] = sin(a->data[get_index(a, i)]);
    }
}

void tensor_cos_cpu(Tensor *a, float *result)
{
    for (int32_t i = 0; i < a->size; i++)
    {
        result[i] = cos(a->data[get_index(a, i)]);
    }
}

void tensor_tanh_cpu(Tensor *a, float *result)
{
    for (int32_t i = 0; i < a->size; i++)
    {
        result[i] = tanh(a->data[get_index(a, i)]);
    }
}

float tensor_sum_cpu(Tensor *a)
{
    float result = 0;
    for (int32_t i = 0; i < a->size; i++)
    {
        result += a->data[get_index(a, i)];
    }
    return result;
}

float tensor_mean_cpu(Tensor *a)
{
    return tensor_sum_cpu(a) / a->size;
}

float tensor_min_cpu(Tensor *a)
{
    float result = a->data[get_index(a, 0)];
    for (int32_t i = 0; i < a->size; i++)
    {
        int32_t j = get_index(a, i);
        if (a->data[j] < result)
        {
            result = a->data[j];
        }
    }
    return result;
}

float tensor_max_cpu(Tensor *a)
{
    float result = a->data[get_index(a, 0)];
    for (int32_t i = 0; i < a->size; i++)
    {
        int32_t j = get_index(a, i);
        if (a->data[j] > result)
        {
            result = a->data[j];
        }
    }
    return result;
}