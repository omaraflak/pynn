#include "hip/hip_runtime.h"
#include "tensor.h"

__device__ uint32_t get_index_in_tensor(Tensor *tensor, uint32_t *indices)
{
    uint32_t index = 0;
    for (uint32_t i = 0; i < tensor->dims; i++)
    {
        index += tensor->stride[i] * indices[i];
    }
    return index;
}

__global__ void matmul_tensor_kernel(Tensor *x, Tensor *y, Tensor *result)
{
    uint32_t x_index = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t x_stride = gridDim.x * blockDim.x;
    uint32_t y_index = blockDim.y * blockIdx.y + threadIdx.y;
    uint32_t y_stride = gridDim.y * blockDim.y;

    uint32_t indices[2];
    uint32_t idx1, idx2, idx3;

    for (uint32_t i = y_index; i < result->shape[0]; i += y_stride)
    {
        for (uint32_t j = x_index; j < result->shape[1]; j += x_stride)
        {
            float tmp = 0;
            for (uint32_t k = 0; k < y->shape[0]; k++)
            {
                indices[0] = i;
                indices[1] = k;
                idx1 = get_index_in_tensor(x, indices);
                indices[0] = k;
                indices[1] = j;
                idx2 = get_index_in_tensor(y, indices);
                tmp += x->data[idx2] * y->data[idx3];
            }

            indices[0] = i;
            indices[1] = j;
            idx1 = get_index_in_tensor(result, indices);
            result->data[idx1] = tmp;
        }
    }
}

extern "C"
{
    Tensor *create_tensor(uint32_t *shape, uint32_t dims)
    {
        Tensor *tensor = new Tensor();
        tensor->dims = dims;
        tensor->size = get_size_from_shape(shape, dims);
        tensor->data = new float[tensor->size];
        tensor->shape = new uint32_t[dims];
        tensor->stride = new uint32_t[dims];
        for (uint32_t i = 0; i < dims; i++)
        {
            tensor->shape[i] = shape[i];
        }
        for (uint32_t i = 0; i < dims; i++)
        {
            tensor->stride[i] = 1;
            for (uint32_t j = i + 1; j < dims; j++)
            {
                tensor->stride[i] *= shape[j];
            }
        }
        return tensor;
    }

    void delete_tensor(Tensor *tensor)
    {
        delete[] tensor->data;
        delete[] tensor->shape;
        delete[] tensor->stride;
        delete tensor;
    }

    Tensor *create_device_tensor(uint32_t *shape, uint32_t dims)
    {
        Tensor *host = create_tensor(shape, dims);
        Tensor *device = to_device(host);
        delete_tensor(host);
        return device;
    }

    void delete_device_tensor(Tensor *tensor)
    {
        Tensor host;
        hipMemcpy(&host, tensor, sizeof(Tensor), hipMemcpyDeviceToHost);
        hipFree(host.data);
        hipFree(host.shape);
        hipFree(host.stride);
        hipFree(tensor);
    }

    uint32_t get_size_from_shape(uint32_t *shape, uint32_t dims)
    {
        uint32_t size = 1;
        for (uint32_t i = 0; i < dims; i++)
        {
            size *= shape[i];
        }
        return size;
    }

    void fill_tensor(Tensor *x, float value)
    {
        for (uint32_t i = 0; i < x->size; i++)
        {
            x->data[i] = value;
        }
    }

    Tensor *to_device(Tensor *tensor)
    {
        float *data;
        uint32_t *shape;
        uint32_t *stride;

        hipMalloc(&data, sizeof(float) * tensor->size);
        hipMalloc(&shape, sizeof(uint32_t) * tensor->dims);
        hipMalloc(&stride, sizeof(uint32_t) * tensor->dims);
        hipMemcpy(data, tensor->data, sizeof(float) * tensor->size, hipMemcpyHostToDevice);
        hipMemcpy(shape, tensor->shape, sizeof(uint32_t) * tensor->dims, hipMemcpyHostToDevice);
        hipMemcpy(stride, tensor->stride, sizeof(uint32_t) * tensor->dims, hipMemcpyHostToDevice);

        Tensor host;
        host.data = data;
        host.shape = shape;
        host.stride = stride;
        host.dims = tensor->dims;
        host.size = tensor->size;

        Tensor *device;
        hipMalloc(&device, sizeof(Tensor));
        hipMemcpy(device, &host, sizeof(Tensor), hipMemcpyHostToDevice);
        return device;
    }

    Tensor *to_host(Tensor *tensor)
    {
        Tensor device;
        hipMemcpy(&device, tensor, sizeof(Tensor), hipMemcpyDeviceToHost);

        Tensor *host = new Tensor();
        host->data = new float[device.size];
        host->shape = new uint32_t[device.dims];
        host->stride = new uint32_t[device.dims];
        host->dims = device.dims;
        host->size = device.size;

        hipMemcpy(host->data, device.data, sizeof(float) * device.size, hipMemcpyDeviceToHost);
        hipMemcpy(host->stride, device.stride, sizeof(uint32_t) * device.dims, hipMemcpyDeviceToHost);
        hipMemcpy(host->shape, device.shape, sizeof(uint32_t) * device.dims, hipMemcpyDeviceToHost);

        return host;
    }

    Tensor *add_tensor(Tensor *a, Tensor *b)
    {
        Tensor *result = create_tensor(a->shape, a->dims);
        for (uint32_t i = 0; i < result->size; i++)
        {
            result->data[i] = a->data[i] + b->data[i];
        }
        return result;
    }

    Tensor *subtract_tensor(Tensor *a, Tensor *b)
    {
        Tensor *result = create_tensor(a->shape, a->dims);
        for (uint32_t i = 0; i < result->size; i++)
        {
            result->data[i] = a->data[i] - b->data[i];
        }
        return result;
    }

    Tensor *multiply_tensor(Tensor *a, Tensor *b)
    {
        Tensor *result = create_tensor(a->shape, a->dims);
        for (uint32_t i = 0; i < result->size; i++)
        {
            result->data[i] = a->data[i] * b->data[i];
        }
        return result;
    }

    Tensor *divide_tensor(Tensor *a, Tensor *b)
    {
        Tensor *result = create_tensor(a->shape, a->dims);
        for (uint32_t i = 0; i < result->size; i++)
        {
            result->data[i] = a->data[i] / b->data[i];
        }
        return result;
    }

    Tensor *matmul_tensor(Tensor *a, Tensor *b)
    {
        uint32_t shape[2] = {a->shape[0], b->shape[1]};
        Tensor *result = create_tensor(shape, 2);
        for (int i = 0; i < shape[0]; i++)
        {
            for (int j = 0; j < shape[1]; j++)
            {
                float tmp = 0;
                for (int k = 0; k < b->shape[0]; k++)
                {
                    tmp += a->data[i * a->shape[0] + k] * b->data[k * b->shape[0] + j];
                }
                result->data[i * shape[0] + j] = tmp;
            }
        }
        return result;
    }

    Tensor *matmul_tensor_gpu(Tensor *tensor1, Tensor *tensor2)
    {
        uint32_t shape[2];
        shape[0] = tensor1->shape[0];
        shape[1] = tensor2->shape[1];

        Tensor *x = to_device(tensor1);
        Tensor *y = to_device(tensor2);
        Tensor *result = create_device_tensor(shape, 2);

        dim3 block_dim(shape[0], shape[1]);
        dim3 grid_dim(1, 1);
        if (shape[0] * shape[1] > 1024)
        {
            block_dim.x = 32;
            block_dim.y = 32;
            grid_dim.x = (shape[1] + block_dim.x - 1) / block_dim.x;
            grid_dim.y = (shape[0] + block_dim.y - 1) / block_dim.y;
        }

        matmul_tensor_kernel<<<grid_dim, block_dim>>>(x, y, result);
        hipDeviceSynchronize();

        Tensor *host = to_host(result);

        delete_device_tensor(x);
        delete_device_tensor(y);
        delete_device_tensor(result);

        return host;
    }
}