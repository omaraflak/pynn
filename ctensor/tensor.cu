#include "hip/hip_runtime.h"
#include "tensor.h"
#include "cpu.h"
#include "gpu.h"
#include <cstring>

uint32_t *_copy_shape(Tensor *tensor)
{
    uint32_t *shape = (uint32_t *)malloc(sizeof(uint32_t) * tensor->dims);
    memcpy(shape, tensor->shape, sizeof(uint32_t) * tensor->dims);
    return shape;
}

uint32_t _get_size_from_shape(uint32_t *shape, uint32_t dims)
{
    uint32_t size = 1;
    for (uint32_t i = 0; i < dims; i++)
    {
        size *= shape[i];
    }
    return size;
}

Tensor *_tensor_create(float *data, uint32_t *shape, uint32_t dims, uint32_t device)
{
    Tensor *tensor = (Tensor *)malloc(sizeof(Tensor));
    tensor->data = data;
    tensor->shape = shape;
    tensor->stride = (uint32_t *)malloc(sizeof(uint32_t) * dims);
    tensor->size = _get_size_from_shape(shape, dims);
    tensor->dims = dims;
    tensor->device = device;
    for (uint32_t i = 0; i < dims; i++)
    {
        tensor->stride[i] = 1;
        for (uint32_t j = i + 1; j < dims; j++)
        {
            tensor->stride[i] *= shape[j];
        }
    }
    return tensor;
}

Tensor *tensor_create(float *data, uint32_t *shape, uint32_t dims)
{
    Tensor *tensor = tensor_create_empty(shape, dims);
    for (uint32_t i = 0; i < tensor->size; i++)
    {
        tensor->data[i] = data[i];
    }
    return tensor;
}

Tensor *tensor_create_empty(uint32_t *shape, uint32_t dims)
{
    uint32_t size = _get_size_from_shape(shape, dims);
    Tensor *tensor = (Tensor *)malloc(sizeof(Tensor));
    tensor->data = (float *)malloc(sizeof(float) * size);
    tensor->shape = (uint32_t *)malloc(sizeof(uint32_t) * dims);
    tensor->stride = (uint32_t *)malloc(sizeof(uint32_t) * dims);
    tensor->size = size;
    tensor->dims = dims;
    tensor->device = 0;
    for (uint32_t i = 0; i < dims; i++)
    {
        tensor->shape[i] = shape[i];
        tensor->stride[i] = 1;
        for (uint32_t j = i + 1; j < dims; j++)
        {
            tensor->stride[i] *= shape[j];
        }
    }
    return tensor;
}

Tensor *tensor_copy(Tensor *tensor)
{
    uint32_t *shape = _copy_shape(tensor);

    float *data;
    if (tensor->device == 0)
    {
        data = (float *)malloc(sizeof(float) * tensor->size);
        memcpy(data, tensor->data, sizeof(float) * tensor->size);
    }
    else
    {
        hipMalloc(&data, sizeof(float) * tensor->size);
        hipMemcpy(data, tensor->data, sizeof(float) * tensor->size, hipMemcpyDeviceToDevice);
    }

    return _tensor_create(data, shape, tensor->dims, tensor->device);
}

void tensor_delete(Tensor *tensor)
{
    if (tensor->device == 0)
    {
        free(tensor->data);
    }
    else
    {
        hipFree(tensor->data);
    }
    free(tensor->shape);
    free(tensor->stride);
    free(tensor);
}

void tensor_cpu_to_gpu(Tensor *tensor)
{
    float *data;
    hipMalloc(&data, sizeof(float) * tensor->size);
    hipMemcpy(data, tensor->data, sizeof(float) * tensor->size, hipMemcpyHostToDevice);
    free(tensor->data);
    tensor->data = data;
    tensor->device = 1;
}

void tensor_gpu_to_cpu(Tensor *tensor)
{
    float *data = (float *)malloc(sizeof(float) * tensor->size);
    hipMemcpy(data, tensor->data, sizeof(float) * tensor->size, hipMemcpyDeviceToHost);
    hipFree(tensor->data);
    tensor->data = data;
    tensor->device = 0;
}

void tensor_fill(Tensor *tensor, float value)
{
    if (tensor->device == 0)
    {
        tensor_fill_cpu(tensor, value);
    }
    else
    {
        tensor_fill_gpu(tensor, value);
    }
}

void tensor_fill_random_uniform(Tensor *tensor, float min, float max)
{
    if (tensor->device == 0)
    {
        tensor_fill_random_uniform_cpu(tensor, min, max);
    }
    else
    {
        tensor_fill_random_uniform_gpu(tensor, min, max);
    }
}

void tensor_fill_random_normal(Tensor *tensor, float mean, float std)
{
    tensor_fill_random_normal_cpu(tensor, mean, std);
}

void tensor_reshape(Tensor *tensor, uint32_t *shape, uint32_t dims)
{
    if (tensor->dims != dims)
    {
        free(tensor->shape);
        free(tensor->stride);
        tensor->dims = dims;
        tensor->shape = (uint32_t *)malloc(sizeof(uint32_t) * dims);
        tensor->stride = (uint32_t *)malloc(sizeof(uint32_t) * dims);
    }
    for (uint32_t i = 0; i < dims; i++)
    {
        tensor->shape[i] = shape[i];
    }
    for (uint32_t i = 0; i < dims; i++)
    {
        tensor->stride[i] = 1;
        for (uint32_t j = i + 1; j < dims; j++)
        {
            tensor->stride[i] *= shape[j];
        }
    }
}

float tensor_get_item(Tensor *tensor, uint32_t *indices)
{
    uint32_t index = 0;
    for (uint32_t i = 0; i < tensor->dims; i++)
    {
        index += tensor->stride[i] * indices[i];
    }
    return tensor->data[index];
}

float tensor_sum(Tensor *tensor)
{
    return tensor_sum_cpu(tensor);
}

float tensor_mean(Tensor *tensor)
{
    return tensor_mean_cpu(tensor);
}

float tensor_min(Tensor *tensor)
{
    return tensor_min_cpu(tensor);
}

float tensor_max(Tensor *tensor)
{
    return tensor_max_cpu(tensor);
}

Tensor *tensor_unary_minus(Tensor *tensor)
{
    uint32_t *shape = _copy_shape(tensor);
    float *data;

    if (tensor->device == 0)
    {
        data = (float *)malloc(sizeof(float) * tensor->size);
        tensor_unary_minus_cpu(tensor, data);
    }
    else
    {
        hipMalloc(&data, sizeof(float) * tensor->size);
        tensor_unary_minus_gpu(tensor, data);
    }
    return _tensor_create(data, shape, tensor->dims, tensor->device);
}

Tensor *tensor_add(Tensor *a, Tensor *b)
{
    uint32_t *shape = _copy_shape(a);
    float *data;

    if (a->device == 0 && b->device == 0)
    {
        data = (float *)malloc(sizeof(float) * a->size);
        tensor_add_cpu(a, b, data);
    }
    else
    {
        hipMalloc(&data, sizeof(float) * a->size);
        tensor_add_gpu(a, b, data);
    }
    return _tensor_create(data, shape, a->dims, a->device);
}

Tensor *tensor_subtract(Tensor *a, Tensor *b)
{
    uint32_t *shape = _copy_shape(a);
    float *data;

    if (a->device == 0 && b->device == 0)
    {
        data = (float *)malloc(sizeof(float) * a->size);
        tensor_subtract_cpu(a, b, data);
    }
    else
    {
        hipMalloc(&data, sizeof(float) * a->size);
        tensor_subtract_gpu(a, b, data);
    }
    return _tensor_create(data, shape, a->dims, a->device);
}

Tensor *tensor_multiply(Tensor *a, Tensor *b)
{
    uint32_t *shape = _copy_shape(a);
    float *data;

    if (a->device == 0 && b->device == 0)
    {
        data = (float *)malloc(sizeof(float) * a->size);
        tensor_multiply_cpu(a, b, data);
    }
    else
    {
        hipMalloc(&data, sizeof(float) * a->size);
        tensor_multiply_gpu(a, b, data);
    }
    return _tensor_create(data, shape, a->dims, a->device);
}

Tensor *tensor_divide(Tensor *a, Tensor *b)
{
    uint32_t *shape = _copy_shape(a);
    float *data;

    if (a->device == 0 && b->device == 0)
    {
        data = (float *)malloc(sizeof(float) * a->size);
        tensor_divide_cpu(a, b, data);
    }
    else
    {
        hipMalloc(&data, sizeof(float) * a->size);
        tensor_divide_gpu(a, b, data);
    }
    return _tensor_create(data, shape, a->dims, a->device);
}

Tensor *tensor_matmul(Tensor *a, Tensor *b)
{
    uint32_t *shape = (uint32_t *)malloc(sizeof(uint32_t) * 2);
    shape[0] = a->shape[0];
    shape[1] = b->shape[1];
    uint32_t size = shape[0] * shape[1];
    float *data;

    if (a->device == 0 && b->device == 0)
    {
        data = (float *)malloc(sizeof(float) * size);
        tensor_matmul_cpu(a, b, data);
    }
    else
    {
        hipMalloc(&data, sizeof(float) * size);
        tensor_matmul_gpu(a, b, data);
    }
    return _tensor_create(data, shape, /* dims=*/2, a->device);
}

Tensor *tensor_broadcast_add(Tensor *tensor, float value)
{
    uint32_t *shape = _copy_shape(tensor);
    float *data;

    if (tensor->device == 0)
    {
        data = (float *)malloc(sizeof(float) * tensor->size);
        tensor_broadcast_add_cpu(tensor, value, data);
    }
    else
    {
        hipMalloc(&data, sizeof(float) * tensor->size);
        tensor_broadcast_add_gpu(tensor, value, data);
    }
    return _tensor_create(data, shape, tensor->dims, tensor->device);
}

Tensor *tensor_broadcast_subtract(Tensor *tensor, float value)
{
    uint32_t *shape = _copy_shape(tensor);
    float *data;

    if (tensor->device == 0)
    {
        data = (float *)malloc(sizeof(float) * tensor->size);
        tensor_broadcast_subtract_cpu(tensor, value, data);
    }
    else
    {
        hipMalloc(&data, sizeof(float) * tensor->size);
        tensor_broadcast_subtract_gpu(tensor, value, data);
    }
    return _tensor_create(data, shape, tensor->dims, tensor->device);
}

Tensor *tensor_broadcast_multiply(Tensor *tensor, float value)
{
    uint32_t *shape = _copy_shape(tensor);
    float *data;

    if (tensor->device == 0)
    {
        data = (float *)malloc(sizeof(float) * tensor->size);
        tensor_broadcast_multiply_cpu(tensor, value, data);
    }
    else
    {
        hipMalloc(&data, sizeof(float) * tensor->size);
        tensor_broadcast_multiply_gpu(tensor, value, data);
    }
    return _tensor_create(data, shape, tensor->dims, tensor->device);
}

Tensor *tensor_broadcast_divide(Tensor *tensor, float value)
{
    uint32_t *shape = _copy_shape(tensor);
    float *data;

    if (tensor->device == 0)
    {
        data = (float *)malloc(sizeof(float) * tensor->size);
        tensor_broadcast_divide_cpu(tensor, value, data);
    }
    else
    {
        hipMalloc(&data, sizeof(float) * tensor->size);
        tensor_broadcast_divide_gpu(tensor, value, data);
    }
    return _tensor_create(data, shape, tensor->dims, tensor->device);
}

Tensor *tensor_broadcast_right_divide(Tensor *tensor, float value)
{
    uint32_t *shape = _copy_shape(tensor);
    float *data;

    if (tensor->device == 0)
    {
        data = (float *)malloc(sizeof(float) * tensor->size);
        tensor_broadcast_right_divide_cpu(tensor, value, data);
    }
    else
    {
        hipMalloc(&data, sizeof(float) * tensor->size);
        tensor_broadcast_right_divide_gpu(tensor, value, data);
    }
    return _tensor_create(data, shape, tensor->dims, tensor->device);
}

Tensor *tensor_power(Tensor *tensor, float power)
{
    uint32_t *shape = _copy_shape(tensor);
    float *data;
    if (tensor->device == 0)
    {
        data = (float *)malloc(sizeof(float) * tensor->size);
        tensor_power_cpu(tensor, power, data);
    }
    else
    {
        hipMalloc(&data, sizeof(float) * tensor->size);
        tensor_power_gpu(tensor, power, data);
    }
    return _tensor_create(data, shape, tensor->dims, tensor->device);
}

Tensor *tensor_exp(Tensor *tensor)
{
    uint32_t *shape = _copy_shape(tensor);
    float *data;
    if (tensor->device == 0)
    {
        data = (float *)malloc(sizeof(float) * tensor->size);
        tensor_exp_cpu(tensor, data);
    }
    else
    {
        hipMalloc(&data, sizeof(float) * tensor->size);
        tensor_exp_gpu(tensor, data);
    }
    return _tensor_create(data, shape, tensor->dims, tensor->device);
}

Tensor *tensor_log(Tensor *tensor)
{
    uint32_t *shape = _copy_shape(tensor);
    float *data;
    if (tensor->device == 0)
    {
        data = (float *)malloc(sizeof(float) * tensor->size);
        tensor_log_cpu(tensor, data);
    }
    else
    {
        hipMalloc(&data, sizeof(float) * tensor->size);
        tensor_log_gpu(tensor, data);
    }
    return _tensor_create(data, shape, tensor->dims, tensor->device);
}

Tensor *tensor_log10(Tensor *tensor)
{
    uint32_t *shape = _copy_shape(tensor);
    float *data;
    if (tensor->device == 0)
    {
        data = (float *)malloc(sizeof(float) * tensor->size);
        tensor_log10_cpu(tensor, data);
    }
    else
    {
        hipMalloc(&data, sizeof(float) * tensor->size);
        tensor_log10_gpu(tensor, data);
    }
    return _tensor_create(data, shape, tensor->dims, tensor->device);
}

Tensor *tensor_logb(Tensor *tensor, float base)
{
    uint32_t *shape = _copy_shape(tensor);
    float *data;
    if (tensor->device == 0)
    {
        data = (float *)malloc(sizeof(float) * tensor->size);
        tensor_logb_cpu(tensor, base, data);
    }
    else
    {
        hipMalloc(&data, sizeof(float) * tensor->size);
        tensor_logb_gpu(tensor, base, data);
    }
    return _tensor_create(data, shape, tensor->dims, tensor->device);
}

Tensor *tensor_sin(Tensor *tensor)
{
    uint32_t *shape = _copy_shape(tensor);
    float *data;
    if (tensor->device == 0)
    {
        data = (float *)malloc(sizeof(float) * tensor->size);
        tensor_sin_cpu(tensor, data);
    }
    else
    {
        hipMalloc(&data, sizeof(float) * tensor->size);
        tensor_sin_gpu(tensor, data);
    }
    return _tensor_create(data, shape, tensor->dims, tensor->device);
}

Tensor *tensor_cos(Tensor *tensor)
{
    uint32_t *shape = _copy_shape(tensor);
    float *data;
    if (tensor->device == 0)
    {
        data = (float *)malloc(sizeof(float) * tensor->size);
        tensor_cos_cpu(tensor, data);
    }
    else
    {
        hipMalloc(&data, sizeof(float) * tensor->size);
        tensor_cos_gpu(tensor, data);
    }
    return _tensor_create(data, shape, tensor->dims, tensor->device);
}
